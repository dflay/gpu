// add two numbers 

#include <stdio.h>
#include <hip/hip_runtime.h> 
 
#include <hiprand/hiprand_kernel.h>  

const int NBLOCK  = 1; 
const int NTHREAD = 1; 

__global__ void add(int a,int b,int *c){
   *c = a + b;
}

int main(void){

   int a = 2;
   int b = 7; 
   int c;
 
   int *c_dev; 

   hipMalloc( (void**)&c_dev,sizeof(int) ); 
   add<<<NBLOCK,NTHREAD>>>(a,b,c_dev);
   hipMemcpy(&c,c_dev,sizeof(int),hipMemcpyDeviceToHost); 

   printf("%d + %d = %d \n",a,b,c); 
   
   hipFree(c_dev); 

   return 0;
}
