// add two vectors  

#include <stdio.h>
#include <hip/hip_runtime.h> 
 
#include <hiprand/hiprand_kernel.h>  

#define N 512 

const int NBLOCK  = 1; 
const int NTHREAD = N; 

__global__ void add(int *a,int *b,int *c){
   int tid = threadIdx.x;  // parse data at this index 
   if(tid<N){
      c[tid] = a[tid] + b[tid];
   } 
}

int main(void){

   int a[N],b[N],c[N];
 
   int *a_dev,*b_dev,*c_dev; 

   int i=0;
   for(i=0;i<N;i++){
      a[i] = -1*i; 
      b[i] = 2*i; 
   }

   hipMalloc( (void**)&a_dev,N*sizeof(int) ); 
   hipMalloc( (void**)&b_dev,N*sizeof(int) ); 
   hipMalloc( (void**)&c_dev,N*sizeof(int) ); 

   hipMemcpy(a_dev,a,N*sizeof(int),hipMemcpyHostToDevice); 
   hipMemcpy(b_dev,b,N*sizeof(int),hipMemcpyHostToDevice); 

   add<<<NBLOCK,NTHREAD>>>(a_dev,b_dev,c_dev);

   hipMemcpy(&c,c_dev,N*sizeof(int),hipMemcpyDeviceToHost); 

   for(i=0;i<N;i++){
      printf("i = %d, %d + %d = %d \n",i,a[i],b[i],c[i]);
   } 
  
   // free(a);
   // free(b);
   // free(c);  
   hipFree(a_dev); 
   hipFree(b_dev); 
   hipFree(c_dev); 

   return 0;
}