#include "hip/hip_runtime.h"
// add two vectors  

#include <stdio.h>
#include <hip/hip_runtime.h> 
#include <hip/hip_runtime.h> 
#include <hiprand/hiprand_kernel.h>  

#define N 512 

const int NBLOCK  = N; 
const int NTHREAD = 1; 

__global__ void add(int a,int b,int *c){
   int tid = blockIdx.x;  // parse data at this index 
   if(tid<N){
      c[tid] = a[tid] + b[tid];
   } 
}

int main(void){

   int a[N],b[N],c[N];
 
   int *a_dev,*b_dev,*c_dev; 

   for(i=0;i<N;i++){
      a[i] = -1*i; 
      b[i] = i*i*i; 
   }

   hipMalloc( (void**)&a_dev,N*sizeof(int) ); 
   hipMalloc( (void**)&b_dev,N*sizeof(int) ); 
   hipMalloc( (void**)&c_dev,N*sizeof(int) ); 

   hipMemcpy(a_dev,a,N*sizeof(int),hipMemcpyHostToDevice); 
   hipMemcpy(b_dev,b,N*sizeof(int),hipMemcpyHostToDevice); 

   add<<<NBLOCK,NTHREAD>>>(a_dev,b_dev,c_dev);

   hipMemcpy(&c,c_dev,sizeof(int),hipMemcpyDeviceToHost); 

   for(i=0;i<N;i++){
      printf("i = %d, %d + %d = %d \n",i,a[i],b[i],c[i]);
   } 
   
   hipFree(a_dev); 
   hipFree(b_dev); 
   hipFree(c_dev); 

   return 0;
}
